#include "hip/hip_runtime.h"
#include <iostream>
#include "rism3d.h"

void RISM3D :: calculate () {
  __global__ void kh(double * dtr, double * dt, double * du);
  __global__ void hnc(double * dtr, double * dt, double * du);
  __global__ void trm1mt(double2 * dguv, double * dtr, double * dt,
                         double * dfr, double qv);
  __global__ void mqvfk(double2 * dguv, double2 * dfk, double qv);
  __global__ void oz(double2 * dhuv, double2 * dguv, double * dx, int natv);
  __global__ void tr(double2 * dguv, double * dtr, double2 * dhuv);

  int ng = ce -> ngrid;

  if (clos == 0) {
    for (int iv = 0; iv < sv -> natv; ++iv) {
      kh <<< g, b >>> (dtr + (iv * ng), dt + (iv * ng), du + (iv * ng));
    }
  } else if (clos == 1) {
    for (int iv = 0; iv < sv -> natv; ++iv) {
      hnc <<< g, b >>> (dtr + (iv * ng), dt + (iv * ng), du + (iv * ng));
    }
  } 

  for (int iv = 0; iv < sv -> natv; ++iv) {
    trm1mt <<< g, b >>> (dguv + (iv * ng), dtr + (iv * ng),
			  dt + (iv * ng), dfr, sv -> qv[iv]);
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    fft -> execute(dguv + (iv * ng), - 1);
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    mqvfk <<< g, b >>> (dguv + (iv * ng), dfk, sv -> qv[iv]);
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    oz <<< g, b >>> (dhuv + (iv * ng), dguv,
		      sv -> dx + (iv * sv -> natv * ng), sv -> natv);
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    fft -> execute(dhuv + (iv * ng), 1);
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    tr <<< g, b >>> (dguv + (iv * ng), dtr + (iv * ng), dhuv + (iv * ng));
  }
} 
