
#include <hip/hip_runtime.h>
__global__ void kh(double * dtr, double * dt, double * du) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  double earg = - du[ip] + dt[ip];
  if (earg >= 0.0) {
    dtr[ip] = 1.0 + earg;
  } else {
    dtr[ip] = exp(earg);
  }
}

__global__ void hnc(double * dtr, double * dt, double * du) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dtr[ip] = exp(- du[ip] + dt[ip]);
}

__global__ void trm1mt(double2 * dguv, double * dtr, double * dt,
		       double * dfr, double qv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x = dtr[ip] - 1.0 - dt[ip] + qv * dfr[ip];
  dguv[ip].y = 0.0;
}

__global__ void pqvfr(double2 * dguv, double * dfr, double qv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x += qv * dfr[ip];
}

__global__ void mqvfk(double2 * dguv, double2 * dfk, double qv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x -= qv * dfk[ip].x;
  dguv[ip].y -= qv * dfk[ip].y;
}

__global__ void oz(double2 * dhuv, double2 * dguv, double * dx, int natv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  unsigned int ngr = blockDim.x * gridDim.x * gridDim.y;
  double hr = 0.0;
  double hi = 0.0;
  for (unsigned int iv = 0; iv < natv; ++iv) {
    unsigned int i = ip + iv * ngr;
    hr += dguv[i].x * dx[i];
    hi += dguv[i].y * dx[i];
  }
  dhuv[ip].x = hr;
  dhuv[ip].y = hi;
}

__global__ void tr(double2 * dguv, double * dtr, double2 * dhuv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x = dtr[ip];
  dtr[ip] = dhuv[ip].x + 1.0 - dguv[ip].x;
  //  dguv[ip].y = 0.0;
}
