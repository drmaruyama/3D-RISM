#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "rism3d.h"
#include "extension.h"

void RISM3D :: iterate() {
  void alloc2D (vector <double *> &, int, int);
  void calloc2D (vector <complex <double> *> &, int, int);

  calloc2D (guv, sv -> natv, ce -> ngrid);
  calloc2D (huv, sv -> natv, ce -> ngrid);
  alloc2D (tuv, sv -> natv, ce -> ngrid);

  hipMalloc(&dguv, ce -> ngrid * sv -> natv * sizeof(double2));
  hipMalloc(&dhuv, ce -> ngrid * sv -> natv * sizeof(double2));
  hipMalloc(&dt, ce -> ngrid * sv -> natv * sizeof(double));
  hipMalloc(&dtr, ce -> ngrid * sv -> natv * sizeof(double));
  hipMalloc(&ds, ce -> grid[1] * ce -> grid[2] * sizeof(double));

  ifstream in_file ;
  in_file.open((fname + exttuv).c_str());
  bool saved = in_file.is_open();
  in_file.close();

  if (saved) {
    read_tuv();
  } else {
    initialize_tuv();
  }

  ma -> initialize (ce, sv);
  fft -> initialize (ce);

  cout << "relaxing 3D UV RISM:" << endl;
  bool conver = false;
  for (int istep = 1; istep <= co -> maxstep; ++istep) {
    calculate();
    double rms = cal_rms ();
    if (rms <= co -> convergence) {
      conver = true;
    } else {
      ma -> calculate (dt, dtr);
    }
    cout << " Step = " << istep << " Reside = " << rms << endl;
    if (co -> ksave > 0 && istep % co -> ksave == 0) {
      write_tuv();
    }
    if (conver) {
      if (co -> ksave != 0) {
	write_tuv();
      }
      break;
    }
  }
  if (!conver) {
    cout << "3D UV RISM: reached limit # of relaxation steps: "
	 << co -> maxstep << endl;
  }
  for (int iv = 0; iv < sv -> natv; ++iv) {
    hipMemcpyAsync(huv[iv], dhuv + (iv * ce -> ngrid), 
	       ce -> ngrid * sizeof(double2), hipMemcpyDefault);
    hipMemcpyAsync(guv[iv], dguv + (iv * ce -> ngrid), 
	       ce -> ngrid * sizeof(double2), hipMemcpyDefault);
  }
  delete ma;
  delete fft;
} 
