#include "hip/hip_runtime.h"
#include "solute.h"

void Solute :: init(int n) {
  num = n;
  q = new double[num];
  sig = new double[num];
  eps = new double[num];
  r = new double[num * 3];
}


void Solute :: setup_cuda() {
  hipMalloc(&dq, num * sizeof(double));
  hipMalloc(&dr, num * sizeof(double3));
  hipMemcpyAsync(dq, q, num * sizeof(double), hipMemcpyDefault);
  hipMemcpyAsync(dr, r, num * sizeof(double3), hipMemcpyDefault);
}
