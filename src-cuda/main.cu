#include <iostream>
#include <fstream>
#include "rism3d.h"

int main (int argc, char * argv[]) {
  RISM3D * system;
  int dn;

  system = new RISM3D;

  if (argc == 1) {
    cout << "No parameter file!" << endl ;
    return (1) ;
  }
  if (argc == 2) {
    cout << "Set device 0" << endl ;
    dn = 0;
  }
  if (argc == 3) {
    dn = atoi(argv[2]);
    cout << "Set device " << dn << endl ;
  }
  if (argc > 3) {
    cout << "Too much arguments!" << endl ;
    return (1) ;
  }

  hipSetDevice(dn);
  system -> initialize(argv[1]);
  system -> iterate();
  system -> output();    

  return(0);
}
